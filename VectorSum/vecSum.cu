#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N (1000000)
#define threads_per_block 1024 

__global__ void vecSum(int *a, int *b, int *c, int n){
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  
  if(id < n){
      c[id] = a[id] + b[id];
  }
}

void vecFill(int *a, int n){
  for(int i = 0; i < n; i++){
      a[i] = rand() % 100000;
  }
}

int main(){
  int *h_a, *h_b, *h_c;
  int *d_a, *d_b, *d_c;
  int size = N*sizeof(int);

  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_b, size);
  hipMalloc((void**)&d_c, size);

  h_a = (int*)malloc(size);
  h_b = (int*)malloc(size);
  h_c = (int*)malloc(size);

  vecFill(h_a, N);
  vecFill(h_b, N);

  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  clock_t gputime = clock();

  vecSum<<<N / threads_per_block, threads_per_block>>>(d_a, d_b, d_c, N);

  hipDeviceSynchronize();

  printf("Time of GPU vector sum for a %d sized vector: %f\n", N, ((double)clock() - gputime) / CLOCKS_PER_SEC);

  hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}